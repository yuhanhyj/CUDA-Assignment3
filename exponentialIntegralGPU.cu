#include "hip/hip_runtime.h"
#include "exponentialIntegralGPU.h"
#include "util_gpu.cuh"

#include <cstdio>

const int row_per_batch = 512;

template <typename T>
__device__ __forceinline__ T exponentialIntegral(const int n, const T x,
                                                 const int maxIterations) {
  const T eulerConstant = static_cast<T>(0.5772156649015329);
  const T epsilon       = static_cast<T>(1.E-30);
  const T bigValue      = static_cast<T>(1.0e100);
  int     i, ii, nm1 = n - 1;
  T       a, b, c, d, del, fact, h, psi, ans = static_cast<T>(0.0);

  if (n < 0 || x < 0 || (x == static_cast<T>(0.0) && ((n == 0) || (n == 1)))) {
    return NAN;
  }

  if (x > static_cast<T>(1.0)) {
    b = x + n;
    c = bigValue;
    d = static_cast<T>(1.0) / b;
    h = d;
    for (i = 1; i <= maxIterations; i++) {
      a = -i * (nm1 + i);
      b += static_cast<T>(2.0);
      d   = static_cast<T>(1.0) / (a * d + b);
      c   = b + a / c;
      del = c * d;
      h *= del;
      if (fabs(del - static_cast<T>(1.0)) <= epsilon) {
        ans = h * exp(-x);
        return ans;
      }
    }
    ans = h * exp(-x);
    return ans;
  } else {
    ans  = (nm1 != 0 ? static_cast<T>(1.0) / nm1 : -log(x) - eulerConstant);
    fact = static_cast<T>(1.0);
    for (i = 1; i <= maxIterations; i++) {
      fact *= -x / i;
      if (i != nm1) {
        del = -fact / (i - nm1);
      } else {
        psi = -eulerConstant;
        for (ii = 1; ii <= nm1; ii++) {
          psi += static_cast<T>(1.0) / ii;
        }
        del = fact * (-log(x) + psi);
      }
      ans += del;
      if (fabs(del) < fabs(ans) * epsilon)
        return ans;
    }
    return ans;
  }

  return ans;
}

template <typename T>
__global__ void exponential_integral_kernel(
    const double a, const double division, const int maxIterations,
    const int blocks_per_col, const int start_row, const int n,
    const int numberOfSamples, T* results_gpu) {
  int       row_idx     = blockIdx.x + start_row;
  const int block_steps = blocks_per_col;
  int       ui          = row_idx + 1;

  if (row_idx < n) {
    for (int i = 0; i < numberOfSamples; i += block_steps) {
      int uj = i + threadIdx.x + 1;
      if (uj <= numberOfSamples) {
        T   x                    = static_cast<T>(a + uj * division);
        int glb_idx              = row_idx * numberOfSamples + uj;
        results_gpu[glb_idx - 1] = exponentialIntegral(ui, x, maxIterations);
      }
    }
  }
}

void exponentialIntegralGpu(const unsigned int numberOfSamples,
                            const unsigned int n, const double a,
                            const double division, const int maxIterations,
                            float* resultsGpu, const int block_size,
                            const int stream_num,
                            float     timings[CUDA_STREAMS_MAX]) {
  hipStream_t streams[stream_num];
  TIME_INIT();

  TIME_START();
  const int blocks_per_col = div_up(n, block_size);
  const int grid_size      = row_per_batch;
  TIME_END();

  TIME_START();
  float* results_gpu = NULL;
  CHECK_CUDA(
      hipMalloc((void**) (&results_gpu), n * numberOfSamples * sizeof(float)));
  TIME_END();

  TIME_START();
  if (stream_num > 1) {
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamCreate(&streams[i]));
    }

    for (int start_row = 0; start_row < n;
         start_row += stream_num * row_per_batch) {
      for (int i = 0; i < stream_num; i++) {
        exponential_integral_kernel<<<grid_size, block_size, 0, streams[i]>>>(
            a, division, maxIterations, blocks_per_col,
            start_row + i * row_per_batch, n, numberOfSamples, results_gpu);
      }
    }
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }
  } else {
    for (int start_row = 0; start_row < n; start_row += row_per_batch) {
      exponential_integral_kernel<<<grid_size, block_size>>>(
          a, division, maxIterations, blocks_per_col, start_row, n,
          numberOfSamples, results_gpu);
    }
  }
  TIME_END();

  TIME_START();
  CHECK_CUDA(hipMemcpy(resultsGpu, results_gpu,
                        n * numberOfSamples * sizeof(float),
                        hipMemcpyDeviceToHost));
  TIME_END();

  CHECK_CUDA(hipFree(results_gpu));
  if (stream_num > 1)
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamDestroy(streams[i]));
    }
  TIME_FINISH();

  return;
}

void exponentialIntegralGpu(const unsigned int numberOfSamples,
                            const unsigned int n, const double a,
                            const double division, const int maxIterations,
                            double* resultsGpu, const int block_size,
                            const int stream_num,
                            float     timings[CUDA_STREAMS_MAX]) {
  hipStream_t streams[stream_num];
  TIME_INIT();

  TIME_START();
  const int blocks_per_col = div_up(n, block_size);
  const int grid_size      = row_per_batch;
  TIME_END();

  TIME_START();
  double* results_gpu = NULL;
  CHECK_CUDA(hipMalloc((void**) (&results_gpu),
                        n * numberOfSamples * sizeof(double)));
  TIME_END();

  TIME_START();
  if (stream_num > 1) {
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamCreate(&streams[i]));
    }
    for (int start_row = 0; start_row < n;
         start_row += stream_num * row_per_batch) {
      for (int i = 0; i < stream_num; i++)
        exponential_integral_kernel<<<grid_size, block_size, 0, streams[i]>>>(
            a, division, maxIterations, blocks_per_col,
            start_row + i * row_per_batch, n, numberOfSamples, results_gpu);
    }
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }
  } else {
    for (int start_row = 0; start_row < n; start_row += row_per_batch) {
      exponential_integral_kernel<<<grid_size, block_size>>>(
          a, division, maxIterations, blocks_per_col, start_row, n,
          numberOfSamples, results_gpu);
    }
  }

  TIME_END();

  TIME_START();
  CHECK_CUDA(hipMemcpy(resultsGpu, results_gpu,
                        n * numberOfSamples * sizeof(double),
                        hipMemcpyDeviceToHost));
  TIME_END();

  CHECK_CUDA(hipFree(results_gpu));
  if (stream_num > 1)
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamDestroy(streams[i]));
    }
  TIME_FINISH();

  return;
}
