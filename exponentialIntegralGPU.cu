#include "hip/hip_runtime.h"
#include "exponentialIntegralGPU.h"
#include "util_gpu.cuh"

#include <cstdio>

const int row_per_batch = 512;

template <typename T>
__device__ __forceinline__ T exponentialIntegral(const int n, const T x,
                                                 const int maxIterations) {
  double eulerConstant = 0.5772156649015329;
  double epsilon       = 1.E-30;
  double bigDouble     = 1.0e100;
  int    i, ii, nm1 = n - 1;
  T      a, b, c, d, del, fact, h, psi, ans = 0.0;

  if (n < 0 || x < 0 || (x == 0.0 && ((n == 0) || (n == 1)))) {
    return NAN;
  }

  if (x > 1.0) {
    b = x + n;
    c = bigDouble;
    d = 1.0 / b;
    h = d;
    for (i = 1; i <= maxIterations; i++) {
      a = -i * (nm1 + i);
      b += 2.0;
      d   = 1.0 / (a * d + b);
      c   = b + a / c;
      del = c * d;
      h *= del;
      if (fabs(del - 1.0) <= epsilon) {
        ans = h * exp(-x);
        return ans;
      }
    }
    ans = h * exp(-x);
    return ans;
  } else {
    ans  = (nm1 != 0 ? 1.0 / nm1 : -log(x) - eulerConstant);
    fact = 1.0;
    for (i = 1; i <= maxIterations; i++) {
      fact *= -x / i;
      if (i != nm1) {
        del = -fact / (i - nm1);
      } else {
        psi = -eulerConstant;
        for (ii = 1; ii <= nm1; ii++) {
          psi += 1.0 / ii;
        }
        del = fact * (-log(x) + psi);
      }
      ans += del;
      if (fabs(del) < fabs(ans) * epsilon)
        return ans;
    }
    return ans;
  }

  return ans;
}

template <typename T>
__device__ T exponentialIntegralDeekSeek(const int n, const T x,
                                         const int maxIterations) {
  const T eulerConstant = static_cast<T>(0.5772156649015329);
  const T epsilon       = static_cast<T>(1.E-30);
  const T bigValue      = static_cast<T>(1.0E+300);
  int     i, ii, nm1 = n - 1;
  T       a, b, c, d, del, fact, h, psi, ans = static_cast<T>(0.0);

  if (n < 0 || x < 0 || (x == static_cast<T>(0.0) && ((n == 0) || (n == 1)))) {
    printf("Bad arguments passed to exponentialIntegral\n");
    return static_cast<T>(-1.0);
  }

  if (n == 0) {
    ans = exp(-x) / x;
  } else {
    if (x > static_cast<T>(1.0)) {
      // Continued fraction method (for x > 1)
      b = x + n;
      c = bigValue;
      d = static_cast<T>(1.0) / b;
      h = d;
      for (i = 1; i <= maxIterations; i++) {
        a = -i * (nm1 + i);
        b += static_cast<T>(2.0);
        d   = static_cast<T>(1.0) / (a * d + b);
        c   = b + a / c;
        del = c * d;
        h *= del;
        if (fabs(del - static_cast<T>(1.0)) <= epsilon) {
          ans = h * exp(-x);
          return ans;
        }
      }
      ans = h * exp(-x);
      return ans;
    } else {
      // Series expansion method (for x <= 1)
      ans  = (nm1 != 0 ? static_cast<T>(1.0) / nm1 : -log(x) - eulerConstant);
      fact = static_cast<T>(1.0);
      for (i = 1; i <= maxIterations; i++) {
        fact *= -x / i;
        if (i != nm1) {
          del = -fact / (i - nm1);
        } else {
          psi = -eulerConstant;
          for (ii = 1; ii <= nm1; ii++) {
            psi += static_cast<T>(1.0) / ii;
          }
          del = fact * (-log(x) + psi);
        }
        ans += del;
        if (fabs(del) < fabs(ans) * epsilon)
          return ans;
      }
    }
  }
  return ans;
}

// template <typename T>
// __device__ T exponentialIntegralChatGPT(const int n, const T x, const int
// maxIterations = 100) { // have problem in line 125
//     const T eulerConstant = static_cast<T>(0.5772156649015329);
//     const T epsilon = static_cast<T>(1.E-30);
//     const T big = std::numeric_limits<T>::max();

//     int i, ii, nm1 = n - 1;
//     T a, b, c, d, del, fact, h, psi, ans = static_cast<T>(0);

//     if (n < 0 || x < static_cast<T>(0) || (x == static_cast<T>(0) && (n == 0
//     || n == 1))) {
//         printf("Bad arguments were passed to the exponentialIntegral function
//         call\n"); return static_cast<T>(0);  // or optionally return NaN or
//         -1
//     }

//     if (n == 0) {
//         ans = exp(-x) / x;
//     } else {
//         if (x > static_cast<T>(1.0)) {
//             b = x + static_cast<T>(n);
//             c = big;
//             d = static_cast<T>(1.0) / b;
//             h = d;
//             for (i = 1; i <= maxIterations; ++i) {
//                 a = -static_cast<T>(i) * static_cast<T>(nm1 + i);
//                 b += static_cast<T>(2.0);
//                 d = static_cast<T>(1.0) / (a * d + b);
//                 c = b + a / c;
//                 del = c * d;
//                 h *= del;
//                 if (fabs(del - static_cast<T>(1.0)) <= epsilon) {
//                     ans = h * exp(-x);
//                     return ans;
//                 }
//             }
//             ans = h * exp(-x);
//             return ans;
//         } else {
//             ans = (nm1 != 0) ? static_cast<T>(1.0) / static_cast<T>(nm1) :
//             -log(x) - eulerConstant; fact = static_cast<T>(1.0); for (i = 1;
//             i <= maxIterations; ++i) {
//                 fact *= -x / static_cast<T>(i);
//                 if (i != nm1) {
//                     del = -fact / static_cast<T>(i - nm1);
//                 } else {
//                     psi = -eulerConstant;
//                     for (ii = 1; ii <= nm1; ++ii) {
//                         psi += static_cast<T>(1.0) / static_cast<T>(ii);
//                     }
//                     del = fact * (-log(x) + psi);
//                 }
//                 ans += del;
//                 if (fabs(del) < fabs(ans) * epsilon) return ans;
//             }
//             return ans;
//         }
//     }
//     return ans;
// }

template <typename T>
__global__ void exponential_integral_kernel(
    const double a, const double division, const int maxIterations,
    const int blocks_per_col, const int start_row, const int n,
    const int numberOfSamples, T* results_gpu) {
  int       row_idx     = blockIdx.x + start_row;
  const int block_steps = blocks_per_col;
  int       ui          = row_idx + 1;

  if (row_idx < n) {
    for (int i = 0; i < numberOfSamples; i += block_steps) {
      int uj = i + threadIdx.x + 1;
      if (uj <= numberOfSamples) {
        double x                 = a + uj * division;
        int    glb_idx           = row_idx * numberOfSamples + uj;
        results_gpu[glb_idx - 1] = exponentialIntegral(ui, x, maxIterations);
        // results_gpu[glb_idx-1] = exponentialIntegralDeekSeek(ui, x,
        // maxIterations); results_gpu[glb_idx-1] =
        // exponentialIntegralChatGPT(ui, x, maxIterations);
      }
    }
  }
}

void exponentialIntegralGpu(const unsigned int numberOfSamples,
                            const unsigned int n, const double a,
                            const double division, const int maxIterations,
                            float* resultsGpu, const int block_size,
                            const int stream_num,
                            float     timings[CUDA_STREAMS_MAX]) {
  hipStream_t streams[stream_num];
  TIME_INIT();

  TIME_START();
  const int blocks_per_col = div_up(n, block_size);
  const int grid_size      = row_per_batch;
  TIME_END();

  TIME_START();
  float* results_gpu = NULL;
  CHECK_CUDA(
      hipMalloc((void**) (&results_gpu), n * numberOfSamples * sizeof(float)));
  TIME_END();

  TIME_START();
  if (stream_num > 1) {
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamCreate(&streams[i]));
    }

    for (int start_row = 0; start_row < n;
         start_row += stream_num * row_per_batch) {
      for (int i = 0; i < stream_num; i++) {
        exponential_integral_kernel<<<grid_size, block_size, 0, streams[i]>>>(
            a, division, maxIterations, blocks_per_col,
            start_row + i * row_per_batch, n, numberOfSamples, results_gpu);
      }
    }
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }
  } else {
    for (int start_row = 0; start_row < n; start_row += row_per_batch) {
      exponential_integral_kernel<<<grid_size, block_size>>>(
          a, division, maxIterations, blocks_per_col, start_row, n,
          numberOfSamples, results_gpu);
    }
  }
  TIME_END();

  TIME_START();
  CHECK_CUDA(hipMemcpy(resultsGpu, results_gpu,
                        n * numberOfSamples * sizeof(float),
                        hipMemcpyDeviceToHost));
  TIME_END();

  CHECK_CUDA(hipFree(results_gpu));
  if (stream_num > 1)
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamDestroy(streams[i]));
    }
  TIME_FINISH();

  return;
}

void exponentialIntegralGpu(const unsigned int numberOfSamples,
                            const unsigned int n, const double a,
                            const double division, const int maxIterations,
                            double* resultsGpu, const int block_size,
                            const int stream_num,
                            float     timings[CUDA_STREAMS_MAX]) {
  hipStream_t streams[stream_num];
  TIME_INIT();

  TIME_START();
  const int blocks_per_col = div_up(n, block_size);
  const int grid_size      = row_per_batch;
  TIME_END();

  TIME_START();
  double* results_gpu = NULL;
  CHECK_CUDA(hipMalloc((void**) (&results_gpu),
                        n * numberOfSamples * sizeof(double)));
  TIME_END();

  TIME_START();
  if (stream_num > 1) {
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamCreate(&streams[i]));
    }
    for (int start_row = 0; start_row < n;
         start_row += stream_num * row_per_batch) {
      for (int i = 0; i < stream_num; i++)
        exponential_integral_kernel<<<grid_size, block_size, 0, streams[i]>>>(
            a, division, maxIterations, blocks_per_col,
            start_row + i * row_per_batch, n, numberOfSamples, results_gpu);
    }
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }
  } else {
    for (int start_row = 0; start_row < n; start_row += row_per_batch) {
      exponential_integral_kernel<<<grid_size, block_size>>>(
          a, division, maxIterations, blocks_per_col, start_row, n,
          numberOfSamples, results_gpu);
    }
  }

  TIME_END();

  TIME_START();
  CHECK_CUDA(hipMemcpy(resultsGpu, results_gpu,
                        n * numberOfSamples * sizeof(double),
                        hipMemcpyDeviceToHost));
  TIME_END();

  CHECK_CUDA(hipFree(results_gpu));
  if (stream_num > 1)
    for (int i = 0; i < stream_num; i++) {
      CHECK_CUDA(hipStreamDestroy(streams[i]));
    }
  TIME_FINISH();

  return;
}
